#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <mutex>

#include <ATen/native/cuda/flashneuron.h>
#include <ATen/Context.h>

// P2P define
#include <queue>
#include <ATen/cuda/CUDAEvent.h>

// Half precision
#include <hip/hip_fp16.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <c10/cuda/CUDACachingAllocator.h>

#define find(n) (32 * (unsigned int)(n / 1024) + (n % 32))
#define mask(n) (0x80000000 >> (unsigned int)((n % 1024) / 32))

// P2P-DSA Setting flag define
#define FN_FLAG_VDNN  (1U << 0)
#define FN_FLAG_FP16  (1U << 1)
#define FN_FLAG_CSR   (1U << 2)
#define FN_FLAG_SSD   (1U << 3)
#define FN_FLAG_TESLA (1U << 4)
#define FN_FLAG_RAID0 (1U << 5)
#define FN_FLAG_DEBUG (1U << 6)
// 7~11 bit will be used for memory manager (device) cudamalloc size
#define FN_MEMSIZE_MASK  (0x00000F80)
// 12~16 bit will be used for memory manager (p2p communication) cudamalloc size
#define FN_P2PSIZE_MASK  (0x0001F000)
#define FN_FLAG_TIMER (1U << 17)
#define FN_MEMSIZE_SHIFT (7)

using namespace at::cuda;
__global__ void double_scale(__half *din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = (double)__half2float(din[idx]);
}

__global__ void float_scale(__half *din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = __half2float(din[idx]);
}

__global__ void zero_insert_double(unsigned int *bit, unsigned int *nz_pos, float* din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = (double)din[count + 0];
      } else {
        dout[idx] = (double)din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

__global__ void zero_insert_float(unsigned int *bit, unsigned int *nz_pos, float* din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0f;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = din[count + 0];
      } else {
        dout[idx] = din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

namespace at { namespace native {

using namespace at::cuda;

FN_memory fn_memorymanager;

/*
typedef struct {
  uint64_t addr;
  uint64_t size;
  uint64_t offs;

  c10::Storage *stor;
  arcp2p_info *info;
  hipStream_t str;

  // Additional information for post task
  // for GPU to SSD case,
  //  - We need to keep GPU memory until transfer is done.
  //  - Release right after process is completed
  //  - Therefore, we need to keep "c10::Storage" class at here
  // for SSD to GPU case,
  //  - We need to keep required data for half-to-float conversion
  //  - These are only for FP16 and CSR case
} req_element;

std::queue<req_element> req_queue;
*/

FN_memory::FN_memory(): global_tensor_id_(0), hard_training(false), relu_thru(false), mapping(false),
    gradient_map_accum(0), weight_accum(0), misc_accum(0), isTimer(false),
    isFN(false), isFP16(false), isCSR(false), isUsingSSD(false), isTesla(false), isDebug(false),
    device_sz(0), max_device(0), p2p_sz(0), max_p2p(0) {

  on_the_fly = 0;
  hipEventCreate(&startEvent);
  hipEventCreate(&endEvent);

  int greatestPriority, leastPriority;
  hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
  hipStreamCreateWithPriority(&fn_stream, hipStreamNonBlocking, leastPriority);

  fp16_ptr_arr = new uint64_t[NUM_TENSOR];
  bit_ptr_arr = new uint64_t[NUM_TENSOR];
  pos_ptr_arr = new uint64_t[NUM_TENSOR];
  resize_arr = new int[NUM_TENSOR];
  numel_arr = new size_t[NUM_TENSOR];
  elem_arr = new int[NUM_TENSOR];
/*
  cpl_flu_ptr_arr = new uint64_t[NUM_TENSOR];
  cpl_pre_ptr_arr = new uint64_t[NUM_TENSOR];
  offset_arr = new uint64_t[NUM_TENSOR];
  dir_arr = new arcp2p_dir[NUM_TENSOR];
*/

  event_arr_d2h = new bool[NUM_TENSOR];
  event_arr_h2d = new bool[NUM_TENSOR];

  for(int i = 0; i < NUM_TENSOR; i++) {
    event_arr_d2h[i] = false;
    event_arr_h2d[i] = false;
    feature_map_accum[i] = 0;
  }

//  memset(dir_arr, 0, sizeof(arcp2p_dir) * NUM_TENSOR);
}

FN_memory::~FN_memory() {
  if (device_sz > 0 && isFN) {
    hipFree(deviceAddr);
    delete[] deviceTable;
    delete[] device_page_map;
    delete[] device_page_map_rev;
  }

  if (p2p_sz > 0 && isFN) {
    hipFree(p2pAddr);
    delete[] p2pTable;
    delete[] p2p_page_map;
  }

  delete[] fp16_ptr_arr;
  delete[] bit_ptr_arr;
  delete[] pos_ptr_arr;
  delete[] resize_arr;
  delete[] numel_arr;
  delete[] elem_arr;
/*
  delete[] cpl_flu_ptr_arr;
  delete[] cpl_pre_ptr_arr;
  delete[] offset_arr;
  delete[] dir_arr;
*/

  delete[] event_arr_d2h;
  delete[] event_arr_h2d;

  if (true == isUsingSSD)
  {
/*
    arcp2p_synchronize(arc_handle);
    if (true == isTesla)
    {
      arcp2p_bar_detach(arc_handle);
    }
    arcp2p_release(arc_handle);
*/
  }
}

void FN_memory::device_malloc(void** gpu_ptr, size_t size) {
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int blkCheck = 0;
  int retryCnt = 0;

  if (device_sz == 0) return;

  if (reqBlk == 0) return;

  devStartBlk = &devBlk_0_4;
  devMaxBlk = max_device;

  while (true) {
    for (int i = *devStartBlk; i < devMaxBlk; i++) {
      blkCheck += 1;
      if (deviceTable[i] != 0) {
        if (deviceTable[i] == 2) {
          std::cout << "Table meet with reverse" << std::endl;
        }
        if (device_page_map[i] == 0) {
          std::cout << "device_page_map[" << i << "] is zero, size: " << size << ", " << blkCheck << ", " << reqBlk << std::endl;
          exit(1);
        }
        i += device_page_map[i] - 1;
        blkCheck = 0; *devStartBlk = i + 1;
        continue;
      }

      if (blkCheck == reqBlk) {
        device_page_map[*devStartBlk] = reqBlk;
        *gpu_ptr = (void* )((size_t)deviceAddr + (*devStartBlk * BLK_SZ));

        for (int i = *devStartBlk; i < *devStartBlk + reqBlk; i++) {
          deviceTable[i] = 1;
        }

        *devStartBlk += reqBlk;
        dev_freeBlk -= reqBlk;

        return;
      }
    }
    blkCheck = 0;
    *devStartBlk = 0;

    if (retryCnt++ > 2) {
      if (isDebug) {
        std::cout << "dev malloc failed: " << (double)size / 1024 / 1024 << ", " << device_occupancy() << std::endl;
      }
      *gpu_ptr = NULL;
      return;
    }
  }
}

void FN_memory::device_malloc_reverse(void** gpu_ptr, size_t size) {
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int blkCheck = 0;
  int retryCnt = 0;

  if (device_sz == 0) return;

  if (reqBlk == 0) return;

  devStartBlk_rev = &devBlk_128_rev;
  devMaxBlk = 0;

  while (true) {
    for (int i = *devStartBlk_rev; i >= devMaxBlk; i--) {
      blkCheck += 1;
      if (deviceTable[i] != 0) {
        if (deviceTable[i] == 1) {
          std::cout << "Table meet at first table" << std::endl;
        }
        if (device_page_map_rev[i] == 0) {
          std::cout << "device_page_map_rev[" << i << "] is zero, size: " << size << ", " << blkCheck << ", " << reqBlk << ", " << deviceTable[i] << std::endl;
          exit(1);
        }
        i -= (device_page_map_rev[i] - 1);
        blkCheck = 0; *devStartBlk_rev = i - 1;
        continue;
      }

      if (blkCheck == reqBlk) {
        device_page_map_rev[*devStartBlk_rev] = reqBlk;
        *gpu_ptr = (void* )((size_t)deviceAddr + ((*devStartBlk_rev - reqBlk + 1) * BLK_SZ));

        for (int i = *devStartBlk_rev; i >= *devStartBlk_rev - reqBlk + 1; i--) {
          deviceTable[i] = 2;
        }

        *devStartBlk_rev -= reqBlk;
        dev_freeBlk -= reqBlk;

        return;
      }
    }
    c10::cuda::CUDACachingAllocator::emptyCache();
    blkCheck = 0;
    *devStartBlk_rev = max_device - 1;

    if (retryCnt++ > 2) {
      std::cout << "dev malloc reverse failed: " << (double)size / 1024 / 1024 << ", " << device_occupancy() << std::endl;
      *gpu_ptr = NULL;
      return;
    }
  }
}

void FN_memory::device_free(void* addr, size_t size) {
  int startBlk = ((size_t)addr - (size_t)deviceAddr) / BLK_SZ;
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);

  if (device_sz == 0) return;

  device_page_map[startBlk] = 0;

  bool reverse = false;
  if (deviceTable[startBlk] == 2) {
    reverse = true;
  }

  if (isDebug)
    std::cout << "device_free addr: " << addr << ", size: " << size << ", reverse: " << reverse << std::endl;

  for (unsigned int i = startBlk; i < startBlk + reqBlk; i++) {
    deviceTable[i] = 0;
  }

  dev_freeBlk += reqBlk;

  if (!reverse) {
    devBlk_0_4 = std::min(devBlk_0_4, startBlk);
  } else {
    devBlk_128_rev = std::max(devBlk_128_rev, startBlk + reqBlk - 1);
  }
}

size_t FN_memory::device_occupancy_size() {
  return dev_freeBlk * BLK_SZ;
}

size_t FN_memory::p2p_occupancy_size() {
  return p2p_freeBlk * BLK_SZ;
}

double FN_memory::device_occupancy() {
  return dev_freeBlk / (double)max_device;
}

double FN_memory::device_occupancy_future(size_t size) {
  unsigned int reqBlk = std::ceil((double)size / (double)BLK_SZ);

  return (dev_freeBlk - reqBlk) / (double)max_device;
}


double FN_memory::p2p_occupancy() {
  return p2p_freeBlk / (double)max_p2p;
}

void FN_memory::p2p_malloc(void** gpu_ptr, size_t size) {
  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int blkCheck = 0;
  int retryCnt = 0;

  if (p2p_sz == 0) return;

  if (reqBlk == 0) return;

/*
  if (isDebug) {
    std::cout << "p2p malloc size test sampling: " << (double)size / 1024 / 1024 << std::endl;
  }
*/

  p2pStartBlk = &p2pBlk_0_4;
  p2pMaxBlk = max_p2p;

  while (true) {
    for (int i = *p2pStartBlk; i < p2pMaxBlk; i++) {
      blkCheck += 1;
      if (p2pTable[i]) {
        if (p2p_page_map[i] == 0) {
          std::cout << "p2p_page_map[" << i << "] is zero, size: " << size << ", " << blkCheck << ", " << reqBlk << std::endl;
          exit(1);
        }
        i += p2p_page_map[i] - 1;
        blkCheck = 0; *p2pStartBlk = i + 1;
        continue;
      }

      if (blkCheck == reqBlk) {
        p2p_page_map[*p2pStartBlk] = reqBlk;
        *gpu_ptr = (void* )((size_t)p2pAddr + (*p2pStartBlk * BLK_SZ));

        for (int i = *p2pStartBlk; i < *p2pStartBlk + reqBlk; i++) {
          p2pTable[i] = true;
        }

        *p2pStartBlk += reqBlk;
        p2p_freeBlk -= reqBlk;

        return;
      }
    }
    blkCheck = 0;

    *p2pStartBlk = 0;

    if (retryCnt++ > 2) {
      std::cout << "p2p malloc failed: " << (double)size / 1024 / 1024 << ", " << p2p_occupancy() << std::endl;
      *gpu_ptr = NULL;
      return;
    }
  }
}

void FN_memory::p2p_free(void* addr, size_t size) {
  int startBlk = ((size_t)addr - (size_t)p2pAddr) / BLK_SZ;
//  int reqBlk = std::ceil((double)size / (double)BLK_SZ);
  int reqBlk = p2p_page_map[startBlk];

  if (p2p_sz == 0) return;

  p2p_page_map[startBlk] = 0;

  for (unsigned int i = startBlk; i < startBlk + reqBlk; i++) {
    p2pTable[i] = false;
  }

  p2p_freeBlk += reqBlk;

  p2pBlk_0_4 = std::min(p2pBlk_0_4, startBlk);
}

void* FN_memory::get_fp16_addr(int tid) {
  return (void *)fp16_ptr_arr[tid];
}

void FN_memory::set_fp16_addr(int tid, uint64_t addr) {
  fp16_ptr_arr[tid] = addr;
}

void* FN_memory::get_device_addr() {
  return p2pAddr;
}

uint64_t FN_memory::get_device_sz() {
  return p2p_sz;
}

void* FN_memory::get_bit_addr(int tid) {
  return (void *)bit_ptr_arr[tid];
}

void FN_memory::set_bit_addr(int tid, uint64_t addr) {
  bit_ptr_arr[tid] = addr;
}

void* FN_memory::get_pos_addr(int tid) {
  return (void *)pos_ptr_arr[tid];
}

void FN_memory::set_pos_addr(int tid, uint64_t addr) {
  pos_ptr_arr[tid] = addr;
}

int FN_memory::get_resize(int tid) {
  return resize_arr[tid];
}

void FN_memory::set_resize(int tid, int resize) {
  resize_arr[tid] = resize;
}

size_t FN_memory::get_numel(int tid) {
  return numel_arr[tid];
}

void FN_memory::set_numel(int tid, size_t numel) {
  numel_arr[tid] = numel;
}

int FN_memory::get_elem(int tid) {
  return elem_arr[tid];
}

void FN_memory::set_elem(int tid, int elem) {
  elem_arr[tid] = elem;
}

/*
void* FN_memory::get_cpl_addr(int tid, arcp2p_dir dir) {
  if (arcp2p_gputossd == dir) {
    return (void *)cpl_flu_ptr_arr[tid];
  } else if (arcp2p_ssdtogpu == dir) {
    return (void *)cpl_pre_ptr_arr[tid];
  } else {
    return nullptr;
  }
}

void FN_memory::set_cpl_addr(int tid, arcp2p_dir dir, void *addr) {
  if (arcp2p_gputossd == dir) {
    cpl_flu_ptr_arr[tid] = (uint64_t)addr;
  } else if (arcp2p_ssdtogpu == dir) {
    cpl_pre_ptr_arr[tid] = (uint64_t)addr;
  }
}

uint64_t* FN_memory::get_offset_ptr(int tid) {
  return &offset_arr[tid];
}

arcp2p_dir FN_memory::get_dir(int tid) {
  return dir_arr[tid];
}

void FN_memory::set_dir(int tid, arcp2p_dir dir) {
  dir_arr[tid] = dir;
}
*/

bool FN_memory::is_timer(void) {
  return isTimer;
}

bool FN_memory::is_fn(void) {
  return isFN;
}

bool FN_memory::is_fp16(void) {
  return isFP16;
}

bool FN_memory::is_csr(void) {
  return isCSR;
}

bool FN_memory::is_using_ssd(void) {
  return isUsingSSD;
}

bool FN_memory::is_debug(void) {
  return isDebug;
}

void FN_memory::Arcp2pSetting(int flags) {
  printf("Arcp2pSetting : 0x%x\n", flags);

  uint64_t device_in_gb;
  device_in_gb = (flags & FN_MEMSIZE_MASK) >> FN_MEMSIZE_SHIFT;
  device_sz = device_in_gb << 30;
  max_device = device_sz / BLK_SZ;

  uint64_t p2p_in_gb;
  p2p_in_gb = (flags & FN_P2PSIZE_MASK) >> 12;
  p2p_sz = p2p_in_gb << 30;
  max_p2p = p2p_sz / BLK_SZ;

  init_4m = ((size_t)(max_device * 0.1));
  init_16m = ((size_t)(max_device * 0.3));
  init_64m = ((size_t)(max_device * 0.7));
  init_128m = ((size_t)(max_device * 0.95));

  devBlk_0_4 = 0;
  devBlk_4_16 = init_4m;
  devBlk_16_64 = init_16m;
  devBlk_64_128 = init_64m;
  devBlk_128 = init_128m;

  devBlk_0_4_rev = init_4m - 1;
  devBlk_4_16_rev = init_16m - 1;
  devBlk_16_64_rev = init_64m - 1;
  devBlk_64_128_rev = init_128m - 1;
  devBlk_128_rev = max_device - 1;


  p2pBlk_0_4 = 0;

  printf("Device memory size = %ld GB\n", device_in_gb);
  printf("P2P memory size = %ld GB\n", p2p_in_gb);

  if (device_in_gb > 0) {
    hipMalloc(&deviceAddr, device_sz);
    deviceTable = new short[max_device];
    memset(deviceTable, 0, sizeof(short) * max_device);

    device_page_map = new unsigned int[max_device];
    for (int i = 0; i < max_device; i++) {
      device_page_map[i] = 0;
    }

    device_page_map_rev = new unsigned int[max_device];
    for (int i = 0; i < max_device; i++) {
      device_page_map_rev[i] = 0;
    }

    dev_freeBlk = (double)max_device;
  }

  if (p2p_in_gb > 0) {
    hipMalloc(&p2pAddr, p2p_sz);
    p2pTable = new bool[max_p2p];
    memset(p2pTable, 0, sizeof(bool) * max_p2p);

    p2p_page_map = new unsigned int[max_p2p];
    for (int i = 0; i < max_p2p; i++) {
      p2p_page_map[i] = 0;
    }

    p2p_freeBlk = (double)max_p2p;
  }

  if (flags & FN_FLAG_TIMER) {
    printf("Timer profiler set\n");
    isTimer = true;
  }

  if (flags & FN_FLAG_VDNN) {
    printf("vDNN flag set\n");
    isFN = true;
  }

  if (flags & FN_FLAG_FP16) {
    printf("FP16 flag set\n");
    isFN = true;
    isFP16 = true;
  }

  if (flags & FN_FLAG_CSR) {
    printf("CSR flag set\n");
    isFN = true;
    isFP16 = true;
    isCSR = true;
  }

  if (flags & FN_FLAG_TESLA) {
    printf("Tesla flag set\n");
    isTesla = true;
  }

  if (flags & FN_FLAG_SSD) {
    printf("SSD flag set\n");
    // [JS] P2P
/*
    isFN = true;
    isUsingSSD = true;
    last_allocated_offset = 0;

    const char *nvme_path_tesla[PATH_LENGTH] = {"0000:65:00.00", "0000:66:00.00"}; // TESLA
    const char *nvme_path_quadro[PATH_LENGTH] = {"0000:85:00.00", ""}; // QUADRO
    const int nvme_cnt = (flags & FN_FLAG_RAID0)?2:1;

    printf("RAID0 flag check, device cnt %d\n", nvme_cnt);

    void* lib_handle;
    if (!(lib_handle = dlopen("/usr/local/lib/libarcp2p.so", RTLD_LAZY))) {
      fprintf(stderr, "%s\n", dlerror());
      return;
    }

    arcp2p_initialize = (arcp2p_type1_fn)dlsym(lib_handle, "ARCP2P_initialize");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_release    = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_release");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_bar_attach = (arcp2p_type3_fn)dlsym(lib_handle, "ARCP2P_bar_attach");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_bar_detach = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_bar_detach");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_transfer   = (arcp2p_type4_fn)dlsym(lib_handle, "ARCP2P_transfer");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_completion = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_completion");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    arcp2p_synchronize = (arcp2p_type2_fn)dlsym(lib_handle, "ARCP2P_synchronize");
    if(dlerror()) { fprintf(stderr, "Error linking\n"); return; }

    if (true == isTesla) {
      arc_handle = arcp2p_initialize(nvme_path_tesla, nvme_cnt);
    } else {
      arc_handle = arcp2p_initialize(nvme_path_quadro, nvme_cnt);
    }
*/
  } else { // if not ssd
    isUsingSSD = false;
  }

  if (flags & FN_FLAG_DEBUG) {
    printf("Debug mode on\n");
    isDebug = true;
    at::globalContext().FNGlobal.turnOnDebugMode();
  } else {
    isDebug = false;
  }
}

/*
// bar attach
int  FN_memory::Arcp2pBarMapping(uint64_t addr, uint64_t size) {
  return arcp2p_bar_attach(arc_handle, addr, size);
}

// submission
void FN_memory::Arcp2pSubmission(uint64_t addr, uint64_t size, uint64_t *p_offs,
    arcp2p_cpl *p_cpl, arcp2p_dir dir, c10::Storage *stor, arcp2p_info *info, hipStream_t str) {
  uint64_t offset, aligned_size;

  const uint64_t prp_align_size = (1UL << 12);
  const uint64_t prp_align_mask = (prp_align_size - 1);

  // align up the size value
  if (size & prp_align_mask) {
    aligned_size = (size + prp_align_size - 1) & (~prp_align_mask);
  } else {
    aligned_size = size;
  }

  if (arcp2p_gputossd == dir) {
    // flush case, need to allocate nvme area
    offset = last_allocated_offset;
    last_allocated_offset = last_allocated_offset + aligned_size;

    *p_offs = offset;
  } else {
    // prefetch case, handle requested nvme offset
    offset = *p_offs;
  }

  req_element req;
  req.addr = addr;
  req.size = aligned_size;
  req.dir = dir;
  req.stor = stor;
  req.info = info;
  req.str = str;

  req.offs = offset;
  req.p_cpl = p_cpl;

  req.p_cpl->requested = true;
  req.p_cpl->arc_handle = arc_handle;

  if (true == isTesla) {
    // directly deliver transfer request to arcp2p library, only for tesla
    arcp2p_transfer(arc_handle, addr, offset, aligned_size, req.p_cpl, dir);
  } else {
    // for quadro, we need to attach bar range before transfer
    // check that queue is empty, else case will be handled at completion function
    if (req_queue.empty()) {
      printf("Transfer directly\n");
      //arcp2p_bar_attach(arc_handle, addr, size);
      // debug code. retry 10 times
      int retrycnt = 0;
      while(ARCP2P_NO_ERROR != arcp2p_bar_attach(arc_handle, addr, size)) {
        retrycnt ++;
        printf("Bar attach failed, retry %d/10\n", retrycnt);
        if (retrycnt >= 10) {
          break;
        }
        arcp2p_bar_detach(arc_handle);
      }
      arcp2p_transfer(arc_handle, addr, offset, aligned_size, req.p_cpl, dir);
    }
  }

  on_the_fly += 1;

  req_queue.push(req);
}

bool FN_memory::Arcp2pReqEmpty() {
  return req_queue.empty();
}
*/

// completion
void FN_memory::Arcp2pCompletion() {

  if(isUsingSSD) {
/*
    // if req_list empty, nothing to do
    if (req_queue.empty()) {
      return;
    }

    // first, run completer of arcp2p, this will update cpl.issued
    arcp2p_completion(arc_handle);

    // we only concern command completion sequentially
    req_element req = req_queue.front();

    if (true == req.p_cpl->issued) {
      // if completed request is ssdtogpu
      // 1. we need to update fetch_loc
      // 2. we should remove loc_element
  
      if (arcp2p_gputossd == req.dir) {
        size_t numel = get_numel(req.info->tid);
        int resize = get_resize(req.info->tid);
        if (isFP16 && (resize > 0)) {
          if (isDebug)
            std::cout << "CSR FP16 mem free tid: " << req.info->tid << ", size: " << sizeof(__half) * resize << ", fp16: " << req.info->ptr << std::endl;
  
          p2p_free(req.info->ptr, sizeof(__half) * resize);
        } else if (isFP16 && (resize == 0)) {
          if (isDebug)
            std::cout << "No CSR FP16 mem free tid: " << req.info->tid << ", size: " << sizeof(__half) * numel << ", fp16: " << req.info->ptr << std::endl;
  
          p2p_free(req.info->ptr, sizeof(__half) * numel);
        } else {
          if (isDebug)
            std::cout << "TODO: Duplicated FP16 mem free tid: " << req.info->tid << ", size: " << req.size << ", fp16: " << req.info->ptr << std::endl;
  
          p2p_free(req.info->ptr, req.size);
        }
  
        event_arr_d2h[req.info->tid] = false;
        delete req.info;
  
        if (false == isFP16)
          delete req.stor;

      } else if (arcp2p_ssdtogpu == req.dir) {
        // [TODO] backend job needed for read done case (ex. notify backward operation that data is ready)
        // [TODO] arcp2p_data would be freed here? or after?
  
        // FP16 & CSR handling
        int resize = get_resize(req.info->tid);
  
        if (isFP16 && (resize > 0)) {
          uint64_t nTPB = req.info->ntpb;
          uint64_t numel = req.info->numel;
  
          size_t bit_elements, pos_elements, pos_elements_before;
          bit_elements = (size_t)((numel + 1024 - 1) / 1024) * 32;
          pos_elements_before = (size_t)((numel + 32 - 1) / 32);
          int count = 0;
          while (pos_elements_before != 0) {
            pos_elements_before = pos_elements_before >> 1;  count++;
          }
          pos_elements = 1 << count;
          
          void* bit = fn_memorymanager.get_bit_addr(req.info->tid);
          void* pos = fn_memorymanager.get_pos_addr(req.info->tid);
          float *nz_dst;
          p2p_malloc((void **)&nz_dst, resize * sizeof(float));
          hipMemsetAsync((void *)nz_dst, 0, resize * sizeof(float), req.str);
  
          float_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half *)req.info->ptr, nz_dst, resize);
//          float_scale<<<(resize + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half *)req.info->ptr, nz_dst, resize);
  
          if (fn_memorymanager.get_elem(req.info->tid) == 8) {
            zero_insert_double<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (double *)req.info->dst, numel);
          } else {
            zero_insert_float<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (float *)req.info->dst, numel);
          }
  
          p2p_free((void *)nz_dst, resize * sizeof(float));
        } else if (isFP16 && (resize == 0)) {
          uint64_t nTPB = req.info->ntpb;
          uint64_t numel = req.info->numel;
  
          if (fn_memorymanager.get_elem(req.info->tid) == 8) {
            double_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half* )req.info->ptr, (double* )req.info->dst, numel);
          } else {
            float_scale<<<(numel + nTPB - 1) / nTPB, nTPB, 0, req.str>>>((__half* )req.info->ptr, (float* )req.info->dst, numel);
          }
        } else {
          hipMemcpyAsync(req.info->dst, req.info->ptr, req.size, hipMemcpyDeviceToDevice, req.str);
        }
  
        event_arr_h2d[req.info->tid] = false;
        delete req.info;
      }
  
      on_the_fly -= 1;
      req.p_cpl->requested = false;
  
      // remove current element
      req_queue.pop();
  
      if (false == isTesla) {
        arcp2p_bar_detach(arc_handle);
  
        // check if next event is pending
        if (!req_queue.empty()) {
          req = req_queue.front();
          printf("schedule next one. quadro only\n");
          //arcp2p_bar_attach(arc_handle, req.addr, req.size);
          // debug code. retry 10 times
          int retrycnt = 0;
          while(ARCP2P_NO_ERROR != arcp2p_bar_attach(arc_handle, req.addr, req.size)) {
            retrycnt ++;
            printf("Bar attach failed, retry %d/10\n", retrycnt);
            if (retrycnt >= 10) {
              break;
            }
            arcp2p_bar_detach(arc_handle);
          }
          arcp2p_transfer(arc_handle, req.addr, req.offs, req.size, req.p_cpl, req.dir);
        }
      }
    }
*/
  }
}

/*
void FN_memory::Arcp2pSynchronize() {
  arcp2p_synchronize(arc_handle);
}
*/

void FN_memory::timeStart() {
  hipEventRecord(startEvent);
}

float FN_memory::timeEnd() {
  hipEventRecord(endEvent);
  hipEventSynchronize(endEvent);
  hipEventElapsedTime(&runTime, startEvent, endEvent);

  return runTime;
}

}}
