#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/native/Copy.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <THC/THC.h>

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif


#include <hip/hip_fp16.h>

#include <thrust/device_ptr.h>
#include <thrust/copy.h>

#include <ATen/native/cuda/flashneuron.h>
#include <c10/cuda/CUDACachingAllocator.h>

#define nTPB 512
#define per_threads 256
#define nthreads 256
#define nblocks 256

#define find(n) (32 * (unsigned int)(n / 1024) + (n % 32))
#define mask(n) (0x80000000 >> (unsigned int)((n % 1024) / 32))


namespace at {
namespace native {

using namespace at::cuda;

__global__ void half_scale(float *din, __half *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = __float2half(din[idx]);
}

__global__ void float_scale(__half *din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = __half2float(din[idx]);
}

__global__ void double_scale(__half *din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize)  dout[idx] = (double)__half2float(din[idx]);
}

__global__ void zero_mask(float *din, unsigned int *bit, unsigned int *pos, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    if (din[idx] != 0.0f) {
      atomicAdd(&bit[find(idx)], mask(idx));
      atomicAdd(&pos[(unsigned int)(idx / 32)], 1);
    }
  }
}

__global__ void pos_first(unsigned int* pos, int asize) {
  int total_idx = nblocks * nthreads;

  for (int j = 0; j < (asize / per_threads / total_idx + 1); j++) {
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if ((global_idx + 1) * per_threads - 1 <= asize) {
      for (int i = 0; i < per_threads; i++) {
        int idx = global_idx * per_threads + i;
        if (idx % per_threads != 0) {
          pos[idx] += pos[idx - 1];
        }
      }
    }
  }
}

__global__ void pos_second(unsigned int* pos, unsigned int* opos, int asize) {
  int total_idx = nblocks * nthreads;

  for (int j = 0; j < (asize / per_threads / total_idx + 1); j++) {
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if ((global_idx + 1) * per_threads - 1 <= asize) {
      unsigned int temp = 0;

      for (int i = 0; i < global_idx; i++) {
        int idx = (i + 1) * per_threads - 1;
        temp += pos[idx];
      }

      for (int i = 0; i < per_threads; i++) {
        int idx = (global_idx) * per_threads + i;
        opos[idx] = pos[idx] + temp;
      }
    }
  }
}

__global__ void zero_insert_double(unsigned int *bit, unsigned int *nz_pos, float* din, double *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = (double)din[count + 0];
      } else {
        dout[idx] = (double)din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

__global__ void zero_insert_float(unsigned int *bit, unsigned int *nz_pos, float* din, float *dout, int dsize) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < dsize) {
    int count = -1;
    if ((unsigned int)(bit[find(idx)] & mask(idx)) > 0) {
      for (int i = (int)(idx / 32) * 32; i < idx + 1; i++) {
        unsigned int mask = bit[find(i)] & mask(i);
        if (mask > 0)  count += 1;
      }
    }

    if (count == -1)  dout[idx] = 0.0f;
    else {
      if ((unsigned int)(idx / 32) == 0) {
        dout[idx] = din[count + 0];
      } else {
        dout[idx] = din[count + nz_pos[(unsigned int)(idx / 32) - 1]];
      }
    }
  }
}

struct is_not_zero {
  __host__ __device__
  bool operator()(const float x) {
    return (x != 0);
  }
};

struct is_not_zero_double {
  __host__ __device__
  bool operator()(const double x) {
    return (x != 0);
  }
};

// device-to-device copy, does type conversion
void copy_device_to_device(TensorIterator& iter, bool non_blocking) {
  int64_t numel = iter.numel();

  // We can memcpy the memory if both tensors have the same type AND both
  // tensors are contiguous after dimension coalescing and reordering.
  bool same_type = iter.dtype(0) == iter.dtype(1);
  bool memcpy_eligible = same_type && iter.is_contiguous();

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  CUDAGuard device_guard(src_device);

  // We always perform the copy on the source device, using the current stream
  // on the source device, and we fully synchronize on both src and dst's
  // current streams for completion of the copy. We have to explicitly do this
  // for non-contig copies. This mimics the behavior of cross-device
  // hipMemcpyAsync on the default stream.
  CUDAStream copy_stream = getCurrentCUDAStream(src_device.index());
  if (src_device != dst_device) {
    // This is a cross-device copy on the src current stream and dst current
    // stream. We perform a two-way barrier between both devices' streams
    // before the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are handled, so
    // that no one is operating on the dst memory when we perform the copy.
    // src waits on dst barrier (src already waits on src)
    CUDAEvent dst_ready;
    device_guard.set_device(dst_device);
    dst_ready.record(getCurrentCUDAStream(dst_device.index()));

    device_guard.set_device(src_device);
    dst_ready.block(copy_stream);
  }

  if (memcpy_eligible) {
    void *dst = iter.data_ptr(0);
    void *src = iter.data_ptr(1);
    size_t size = numel * iter.element_size(0);
    if (src != dst || src_device != dst_device) {
      // Perform the copy
      AT_CUDA_CHECK(hipMemcpyAsync(
          dst, src, size,
          hipMemcpyDeviceToDevice,
          copy_stream));
    }
  } else {
    auto dtype = iter.dtype(0);
    if (isQIntType(dtype)) {
      AT_DISPATCH_QINT_TYPES(dtype, "copy_", [&] {
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          kHalf, kBool, kBFloat16, dtype, "copy_", [&] {
            gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
          });
    }
  }

  if (src_device != dst_device) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on src_device, record stream event
    CUDAEvent src_ready;
    src_ready.record(copy_stream);

    device_guard.set_device(dst_device);
    src_ready.block(getCurrentCUDAStream(dst_device.index()));
  }

  AT_CUDA_CHECK(hipGetLastError());
}

static bool copy_requires_temporaries(TensorIterator& iter, bool p2p_enabled) {
  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  if (dst_device == src_device) {
    // We never require temporaries for copies on the same GPU.
    TORCH_INTERNAL_ASSERT(dst_device.is_cuda() && src_device.is_cuda());
    return false;
  }

  bool same_dtype = iter.dtype(0) == iter.dtype(1);
  if (same_dtype && iter.is_contiguous()) {
    // Contiguous same-dtype copies can always use hipMemcpyAsync
    return false;
  } else if (dst_device.is_cuda() && src_device.is_cuda()) {
    // Copies between GPUs can use the copy kernel if P2P is supported
    return !p2p_enabled;
  } else {
    // The remaining cases require temporaries. For example, this includes
    // non-contiguous copies between CPU and GPU.
    return true;
  }
}

static bool maybe_enable_p2p_access(Device dst_device, Device src_device) {
  if (dst_device.is_cpu() || src_device.is_cpu()) {
    return false;
  }
  return THCState_getPeerToPeerAccess(
        globalContext().getTHCState(), src_device.index(), dst_device.index());
}

static void copy_kernel_cuda(TensorIterator& iter, bool non_blocking) {
  AT_ASSERT(iter.ntensors() == 2);

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  // Enable p2p access between devices. (No-op if it involves the CPU)
  bool p2p_enabled = maybe_enable_p2p_access(dst_device, src_device);

  if (copy_requires_temporaries(iter, p2p_enabled)) {
    // NB: this involves recursive calls to copy. Be careful that those copies
    // don't require temporaries or you will cause an infinite recursion!
    auto& dst = iter.tensor(0);
    Tensor dst_contig;
    Tensor src_contig;

    // Type conversions are performed on the CPU for CPU-GPU copies and on
    // the src device for GPU-GPU copies.
    if (iter.device_type(0) == kCUDA) {
      dst_contig = dst.is_contiguous() ? dst : at::empty_like(dst, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).to(iter.dtype(0)).expand_as(dst).contiguous();
    } else {
      bool same_type = iter.dtype(0) == iter.dtype(1);
      dst_contig = (dst.is_contiguous() && same_type) ? dst : at::empty_like(dst, iter.dtype(1), LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).expand_as(dst).contiguous();
    }

    // perform a same-dtype copy on contiguous tensors
    TORCH_INTERNAL_ASSERT(dst_contig.sizes().equals(src_contig.sizes()));
    TORCH_INTERNAL_ASSERT(dst_contig.scalar_type() == src_contig.scalar_type());
    dst_contig.copy_(src_contig, non_blocking);

    // if necessary, copy back into dst
    if (!dst_contig.is_same(dst)) {
      TORCH_INTERNAL_ASSERT(dst_contig.device() == dst.device());
      dst.copy_(dst_contig, non_blocking);
    }
    return;
  }

  // Copy on GPU (or between GPUs)
  if (dst_device.is_cuda() && src_device.is_cuda()) {
    copy_device_to_device(iter, non_blocking);
    return;
  }

  // Copy between CPU and GPU
  cuda::OptionalCUDAGuard device_guard;
  hipMemcpyKind kind;
  if (dst_device.is_cuda() && src_device.is_cpu()) {
    device_guard.set_device(dst_device);
    kind = hipMemcpyHostToDevice;
  } else if (dst_device.is_cpu() && src_device.is_cuda()) {
    device_guard.set_device(src_device);
    kind = hipMemcpyDeviceToHost;
  } else {
    TORCH_INTERNAL_ASSERT(false, "unsupported devices in GPU copy_()");
  }

  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  int64_t nbytes = iter.numel() * iter.element_size(0);
  CUDAStream stream = getCurrentCUDAStream();

  if (non_blocking) {
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    void* ptr = (dst_device == kCPU ? dst : src);
    AT_CUDA_CHECK(THCCachingHostAllocator_recordEvent(ptr, stream));
  } else {
#if HIP_VERSION >= 301
    AT_CUDA_CHECK(hipMemcpyWithStream(dst, src, nbytes, kind, stream));
#else
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    AT_CUDA_CHECK(hipStreamSynchronize(stream));
#endif
  }
}

static void FN_copy_kernel_cuda(TensorIterator& iter, bool non_blocking, int tid, bool is_csr) {
  /* SSD flag only concerned if 'dir' information is properly saved at fn_memorymanager
   * only this case is from offload and prefetch.
   */
  // arcp2p_dir dir = fn_memorymanager.get_dir(tid);
  // bool ssd_flag  = fn_memorymanager.is_using_ssd() && (dir != arcp2p_unused);
  bool ssd_flag  = false;

  /* fp16 & csr option is delivered by flag setting
   * Note. FP16 should be set when csr is set.
   *       So we don't case about FP16=false & CSR=true case
   */
  bool fp16_flag = fn_memorymanager.is_fp16();
  bool csr_flag  = fn_memorymanager.is_csr();

  // Clear dir value, to avoid confusion
  // fn_memorymanager.set_dir(tid, arcp2p_unused);

  AT_ASSERT(iter.ntensors() == 2);

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  // Enable p2p access between devices. (No-op if it invovles the CPU)
  bool p2p_enabled = maybe_enable_p2p_access(dst_device, src_device);

  // Copy between CPU and GPU
  cuda::OptionalCUDAGuard device_guard;
  hipMemcpyKind kind;
  if (dst_device.is_cuda() && src_device.is_cpu()) {
    device_guard.set_device(dst_device);
    kind = hipMemcpyHostToDevice;
    fn_memorymanager.event_arr_h2d[tid] = true;
  } else if (dst_device.is_cpu() && src_device.is_cuda()) {
    device_guard.set_device(src_device);
    kind = hipMemcpyDeviceToHost;
    fn_memorymanager.event_arr_d2h[tid] = true;
  } else {
    TORCH_INTERNAL_ASSERT(false, "unsupported devices in GPU copy_()");
  }

  uint64_t p2p_addr = 0, p2p_size = 0;

  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  int64_t nbytes = iter.numel() * iter.element_size(0);
  hipStream_t stream = at::native::fn_memorymanager.fn_stream;

  fn_memorymanager.set_elem(tid, iter.element_size(0));

  if (true == ssd_flag) {
/*
    if (!fn_memorymanager.mapping) {
      // [TODO] this should be called only for Tesla option enabled
      void* deviceAddr = fn_memorymanager.get_device_addr();
      uint64_t deviceSz = fn_memorymanager.get_device_sz();
      fn_memorymanager.Arcp2pBarMapping((uint64_t)deviceAddr, deviceSz);
      fn_memorymanager.mapping = true;
    }
*/
  }

  size_t bit_elements, pos_elements, pos_elements_before;

  if (csr_flag) {
    bit_elements = (size_t)((iter.numel() + 1024 - 1) / 1024) * 32;
    pos_elements_before = (size_t)((iter.numel() + 32 - 1) / 32);
    int count = 0;
    while (pos_elements_before != 0) {
      pos_elements_before = pos_elements_before >> 1;  count++;
    }
    pos_elements = 1 << count;
  }

  if (kind == hipMemcpyDeviceToHost) {
    if (iter.element_size(0) >= 4) {
      if (csr_flag && is_csr) {
        void *fp16, *bit, *pos;
        fn_memorymanager.p2p_malloc(&bit, sizeof(unsigned int) * bit_elements);
        fn_memorymanager.p2p_malloc(&pos, sizeof(unsigned int) * pos_elements);

        fn_memorymanager.set_bit_addr(tid, (uint64_t)bit);
        fn_memorymanager.set_pos_addr(tid, (uint64_t)pos);

        unsigned int *nz_pos;
        fn_memorymanager.p2p_malloc((void **)&nz_pos, pos_elements * sizeof(unsigned int));

        hipMemsetAsync((void *)bit, 0, sizeof(unsigned int) * bit_elements, stream);
        hipMemsetAsync((void *)pos, 0, sizeof(unsigned int) * pos_elements, stream);
        hipMemsetAsync((void *)nz_pos, 0, sizeof(unsigned int) * pos_elements, stream);

        void *nz_src;
        if (iter.element_size(0) == 8) {
          fn_memorymanager.p2p_malloc((void **)&nz_src, iter.numel() * sizeof(double));
          hipMemsetAsync((void *)nz_src, 0, sizeof(double) * iter.numel(), stream);
          thrust::device_ptr<double> dA_V((double *)src);
          thrust::device_ptr<double> dA_R((double *)nz_src);
          thrust::copy_if(dA_V, dA_V + iter.numel(), dA_R, is_not_zero_double());
        } else {
          fn_memorymanager.p2p_malloc((void **)&nz_src, iter.numel() * sizeof(float));
          hipMemsetAsync((void *)nz_src, 0, sizeof(float) * iter.numel(), stream);
          thrust::device_ptr<float> dA_V((float *)src);
          thrust::device_ptr<float> dA_R((float *)nz_src);
          thrust::copy_if(dA_V, dA_V + iter.numel(), dA_R, is_not_zero());
        }

        zero_mask<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((float *)src, (unsigned int *)bit, nz_pos, iter.numel());

        pos_first<<<nblocks, nthreads, 0, stream>>>(nz_pos, pos_elements);
        pos_second<<<nblocks, nthreads, 0, stream>>>(nz_pos, (unsigned int*)pos, pos_elements);

        int resize = 0;

        hipMemcpyAsync((void *)&resize, (void *)((size_t)pos + sizeof(unsigned int) * (pos_elements - 1)),
            sizeof(int), hipMemcpyDeviceToHost, stream);

        fn_memorymanager.p2p_malloc(&fp16, sizeof(__half) * resize);
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);

        half_scale<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((float *)nz_src, (__half *)fp16, resize);

        fn_memorymanager.set_resize(tid, resize);
        fn_memorymanager.set_numel(tid, iter.numel());

        if (true == ssd_flag) {
/*
          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)(resize * sizeof(__half));
*/
        } else {
          AT_CUDA_CHECK(hipMemcpyAsync(dst, fp16, resize * sizeof(__half), kind, stream));

          fn_memorymanager.p2p_free(fp16, resize * sizeof(__half));
          fn_memorymanager.event_arr_d2h[tid] = false;
        }

        fn_memorymanager.p2p_free((void *)nz_pos, pos_elements * sizeof(unsigned int));
        fn_memorymanager.p2p_free((void *)nz_src, iter.numel() * sizeof(float));
      } else if (fp16_flag) {
        // this case include both cases
        // 1. csr_flag==true && is_csr==false (csr_flag==true always guarantee fp16_flag==true)
        // 2. csr_flag==false && fp16_flag==true

        // keep print message for debug purpose
        void *fp16;
        fn_memorymanager.p2p_malloc(&fp16, sizeof(__half) * iter.numel());
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);

        half_scale<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((float *)src, (__half *)fp16, iter.numel());

        fn_memorymanager.set_resize(tid, 0); // [TODO] slight hack code, we will distinguish CSR / FP16 by resize value
        fn_memorymanager.set_numel(tid, iter.numel());

        if (true == ssd_flag) {
/*
          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)(iter.numel() * sizeof(__half));
*/
        } else {

          AT_CUDA_CHECK(hipMemcpyAsync(dst, fp16, sizeof(__half) * iter.numel(), kind, stream));

          fn_memorymanager.p2p_free(fp16, iter.numel() * sizeof(__half));
          fn_memorymanager.event_arr_d2h[tid] = false;
        }
      } else { // false == csr_flag && false == fp16_flag
        if (true == ssd_flag) {
/*
          // TODO Need to malloc src ptr to BAR attached region
          void *fp16;
          fn_memorymanager.p2p_malloc(&fp16, nbytes);
          fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);
          fn_memorymanager.set_numel(tid, (size_t)nbytes);
          fn_memorymanager.set_resize(tid, -1); // [TODO] slight hack code, we will distinguish CSR / FP16 by resize value
          AT_CUDA_CHECK(hipMemcpyAsync(fp16, src, nbytes, hipMemcpyDeviceToDevice, stream));

          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)nbytes;
*/
        } else {

          fn_memorymanager.set_resize(tid, -1); // [TODO] slight hack code, we will distinguish CSR / FP16 by resize value
          fn_memorymanager.set_numel(tid, iter.numel());
          fn_memorymanager.set_fp16_addr(tid, (uint64_t)NULL);

          AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));

          fn_memorymanager.event_arr_d2h[tid] = false;
        }
      }
    } else { // Non double or float
      if (true == ssd_flag) {
/*
        // TODO Need to malloc src ptr to BAR attached region
        void *fp16;
        fn_memorymanager.p2p_malloc(&fp16, nbytes);
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);
        fn_memorymanager.set_resize(tid, -1); // [TODO] slight hack code, we will distinguish CSR / FP16 by resize value
        fn_memorymanager.set_numel(tid, (size_t)nbytes);
        AT_CUDA_CHECK(hipMemcpyAsync(fp16, src, nbytes, hipMemcpyDeviceToDevice, stream));

        p2p_addr = (uint64_t)fp16;
        p2p_size = (uint64_t)nbytes;
*/
      } else {
        fn_memorymanager.set_resize(tid, -1); // [TODO] slight hack code, we will distinguish CSR / FP16 by resize value
        fn_memorymanager.set_numel(tid, iter.numel());
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)NULL);

        AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));

        fn_memorymanager.event_arr_d2h[tid] = false;
      }
    }
  }

  if (kind == hipMemcpyHostToDevice) {
    if (iter.element_size(0) >= 4) {
      if (csr_flag && is_csr) {
        void* bit = fn_memorymanager.get_bit_addr(tid);
        void* pos = fn_memorymanager.get_pos_addr(tid);

        int resize = fn_memorymanager.get_resize(tid);

        void* fp16;
        fn_memorymanager.p2p_malloc(&fp16, sizeof(__half) * resize);
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);

        if (ssd_flag) {
/*
          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)(resize * sizeof(__half));
          // [JS] all backend job will be called at Arcp2pCompletion
*/
        } else {
          float *nz_dst;
          fn_memorymanager.p2p_malloc((void **)&nz_dst, resize * sizeof(float));
          hipMemsetAsync((void *)nz_dst, 0, resize * sizeof(float), stream);

          AT_CUDA_CHECK(hipMemcpyAsync(fp16, src, resize * sizeof(__half), kind, stream));

          float_scale<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((__half *)fp16, nz_dst, resize);
//          float_scale<<<(resize + nTPB - 1) / nTPB, nTPB, 0, stream>>>((__half *)fp16, nz_dst, resize);

          if (iter.element_size(0) == 8) {
            zero_insert_double<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (double *)dst, iter.numel());
          } else {
            zero_insert_float<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((unsigned int*)bit, (unsigned int*)pos, nz_dst, (float *)dst, iter.numel());
          }

          fn_memorymanager.p2p_free((void *)nz_dst, resize * sizeof(float));
        }
      } else if (fp16_flag) {
        // keep print message for debug purpose
        void* fp16;
        fn_memorymanager.p2p_malloc(&fp16, sizeof(__half) * iter.numel());
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);
        fn_memorymanager.set_numel(tid, iter.numel());
        fn_memorymanager.set_resize(tid, 0);

        if (ssd_flag) {
/*
          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)(iter.numel() * sizeof(__half));
*/
        } else {
          AT_CUDA_CHECK(hipMemcpyAsync(fp16, src, iter.numel() * sizeof(__half), kind, stream));
          if (iter.element_size(0) == 8) {
            double_scale<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((__half* )fp16, (double*)dst, iter.numel());
          } else {
            float_scale<<<(iter.numel() + nTPB - 1) / nTPB, nTPB, 0, stream>>>((__half* )fp16, (float*)dst, iter.numel());
          }

          if (at::globalContext().FNGlobal.isOnDemand()) {
            hipStreamSynchronize(stream);
            fn_memorymanager.event_arr_h2d[tid] = false;
          }
        }
      } else {
        if (true == ssd_flag) {
/*
          void* fp16;
          fn_memorymanager.p2p_malloc(&fp16, nbytes);
          fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);

          p2p_addr = (uint64_t)fp16;
          p2p_size = (uint64_t)nbytes;
*/
        } else {

          fn_memorymanager.set_fp16_addr(tid, (uint64_t)NULL);

          AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));

          if (at::globalContext().FNGlobal.isOnDemand()) {
            hipStreamSynchronize(stream);
            fn_memorymanager.event_arr_h2d[tid] = false;
          }
        }
      }
    } else {
      if (true == ssd_flag) {
/*
        void* fp16;
        fn_memorymanager.p2p_malloc(&fp16, nbytes);
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)fp16);

        p2p_addr = (uint64_t)fp16;
        p2p_size = (uint64_t)nbytes;
*/
      } else {
        fn_memorymanager.set_fp16_addr(tid, (uint64_t)NULL);

        AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
        if (at::globalContext().FNGlobal.isOnDemand()) {
          hipStreamSynchronize(stream);
          fn_memorymanager.event_arr_h2d[tid] = false;
        }
      }
    }
  }

  if (ssd_flag) {
/*
    uint64_t *p_offs = fn_memorymanager.get_offset_ptr(tid);
    arcp2p_cpl *p_cpl = (arcp2p_cpl *)fn_memorymanager.get_cpl_addr(tid, dir);

    if (arcp2p_gputossd == dir) {
      c10::Storage *stor = nullptr;

      if (false == fp16_flag) {
        stor = new c10::Storage;
        *stor = iter.tensor(1).storage();
      }

      arcp2p_info *info = nullptr;

      info = new arcp2p_info;
      info->tid = (uint64_t)tid;
      info->ptr = fn_memorymanager.get_fp16_addr(tid);

      fn_memorymanager.Arcp2pSubmission(p2p_addr, p2p_size, p_offs, p_cpl, dir, stor, info, stream);
      fn_memorymanager.Arcp2pCompletion(false);
    } else if (arcp2p_ssdtogpu == dir) {
      arcp2p_info *info = nullptr;

      info = new arcp2p_info;
      info->tid = (uint64_t)tid;
      info->numel = (uint64_t)iter.numel();
      info->ntpb = nTPB;
      info->dst = iter.data_ptr(0);
      info->src = iter.data_ptr(1);
      info->ptr = fn_memorymanager.get_fp16_addr(tid);

      fn_memorymanager.Arcp2pSubmission(p2p_addr, p2p_size, p_offs, p_cpl, dir, nullptr, info, stream);
      fn_memorymanager.Arcp2pCompletion(false);
    }
*/
  }
}

REGISTER_DISPATCH(copy_stub, &copy_kernel_cuda);
REGISTER_DISPATCH(FN_copy_stub, &FN_copy_kernel_cuda);

} // namespace native
} // namespace at
