#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

#ifdef USE_CUSOLVER

namespace at {
namespace native {

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_getrf_ptr, int* info_getrs_ptr, int n, int lda) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, info_getrf_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, self_inv_ptr, lda, info_getrs_ptr);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  const int lda = std::max<int>(1, n);

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto infos_getrf_data = infos_getrf.data_ptr<int>();
  auto infos_getrs_data = infos_getrs.data_ptr<int>();

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  // Heuristic: For small batch size or large matrix size, we use for-loop to iterate over the batches instead of 
  //            calling the batched cublas routine.
  if (batch_size <= 8 || /* batch_size > 8 && */ n >= 512) {
    for (int64_t i = 0; i < batch_size; i++) {
      auto dataPtr = allocator.allocate(sizeof(int) * lda);
      int* pivot = reinterpret_cast<int*>(dataPtr.get());

      int* infos_getrf_working_ptr = &infos_getrf_data[i];
      int* infos_getrs_working_ptr = &infos_getrs_data[i];

      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, infos_getrf_working_ptr, infos_getrs_working_ptr, n, lda);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<int64_t>(self_data),
      reinterpret_cast<int64_t>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<int64_t>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<int64_t>(self_inv_data),
      reinterpret_cast<int64_t>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<int64_t>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    auto dataPtr = allocator.allocate(sizeof(int)*batch_size*lda);
    int* ipiv_array = reinterpret_cast<int*>(dataPtr.get());

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, infos_getrf_data, batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()), lda, infos_getrs_data, batch_size);
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  int lda = std::max<int>(1, n);

  Tensor ipiv = at::empty({lda}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), infos_getrf.data_ptr<int>(), infos_getrs.data_ptr<int>(), n, lda);
}

// This is a type dispatching helper function for 'apply_batched_inverse_lib' and 'apply_single_inverse_lib'
Tensor& _linalg_inv_out_helper_cuda_lib(Tensor& result, Tensor& infos_getrf, Tensor& infos_getrs) {
  // assuming result is in column major order and contains the matrices to invert
  Tensor input_working_copy = cloneBatchedColumnMajor(result);

  // for getrf + getrs (cusolver path)
  // result should be filled with identity matrices
  result.zero_();
  result.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);

  const int batch_size = cuda_int_cast(batchCount(result), "batchCount");

  if (result.dim() > 2) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        input_working_copy, result, infos_getrf, infos_getrs);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(input_working_copy, result, infos_getrf, infos_getrs);
    });
  }

  return result;
}

// entrance of calculations of `inverse` using cusolver getrf + getrs, cublas getrfBatched + getriBatched
Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos_getrf = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  } else {
    Tensor infos_getrf = at::zeros({1}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({1}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  }

  return self_inv_working_copy;
}

// call cusolver gesvdj function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  for(int i = 0; i < batchsize; i++){
    // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
    hipsolverGesvdjInfo_t gesvdj_params;
    TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));

    auto handle = at::cuda::getCurrentCUDASolverDnHandle();
    auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
    at::cuda::solver::gesvdj<scalar_t>(
      handle, jobz, /*econ=*/ some ? 1 : 0, m, n,
      self_data + i * self_stride,
      lda,
      S_data + i * S_stride,
      U_data + i * U_stride,
      lda,
      VT_data + i * VT_stride,
      ldvt,
      infos.data_ptr<int>() + i,
      gesvdj_params
    );

    TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
  }
}

// wrapper around _apply_svd_lib_gesvdj that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdj", [&] {
    _apply_svd_lib_gesvdj<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv, some);
  });
}

// call cusolver gesvdj batched function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  TORCH_INTERNAL_ASSERT(m <= 32 && n <= 32, "gesvdjBatched requires both matrix dimensions not greater than 32, but got "
                        "m = ", m, " n = ", n);

  // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
  hipsolverGesvdjInfo_t gesvdj_params;
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));
  TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, 1));

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
  at::cuda::solver::gesvdjBatched<scalar_t>(
    handle, jobz, m, n, self_data, lda, S_data, U_data, lda, VT_data, ldvt,
    infos.data_ptr<int>(), gesvdj_params, batchsize
  );

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

// wrapper around _apply_svd_lib_gesvdjBatched that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdjBatched", [&] {
    _apply_svd_lib_gesvdjBatched<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv);
  });
}

// entrance of calculations of `svd` using cusolver gesvdj and gesvdjBatched
std::tuple<Tensor, Tensor, Tensor> _svd_helper_cuda_lib(const Tensor& self, bool some, bool compute_uv) {
  const int64_t batch_size = batchCount(self);
  at::Tensor infos = at::zeros({batch_size}, self.options().dtype(at::kInt));
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  const int64_t k = std::min(m, n);

  Tensor U_working_copy, S_working_copy, VT_working_copy;
  std::tie(U_working_copy, S_working_copy, VT_working_copy) = \
    _create_U_S_VT(self, some, compute_uv, /* svd_use_cusolver = */ true);
  // U, S, V working copies are already column majored now

  // heuristic for using `gesvdjBatched` over `gesvdj`
  if (m <= 32 && n <= 32 && batch_size > 1 && (!some || m == n)) {
    apply_svd_lib_gesvdjBatched(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv);
  } else {
    apply_svd_lib_gesvdj(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv, some);
  }

  // A device-host sync will be performed.
  batchCheckErrors(infos, "svd_cuda");

  if (!compute_uv) {
    VT_working_copy.zero_();
    U_working_copy.zero_();
  }

  if (some) {
    VT_working_copy = VT_working_copy.narrow(-2, 0, k);
  }

  // so far we have computed VT, but torch.svd returns V instead. Adjust accordingly.
  VT_working_copy.transpose_(-2, -1);
  return std::make_tuple(U_working_copy, S_working_copy, VT_working_copy);
}

}} // namespace at::native

#endif  // USE_CUSOLVER
