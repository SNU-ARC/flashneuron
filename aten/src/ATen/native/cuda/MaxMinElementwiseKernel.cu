#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void maximum_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a || b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::max(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        if (a != a) {
          return a;
        } else if (b != b) {
          return b;
        } else {
          return ::max(a, b);
        }
      });
    });
  }
}

void minimum_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "minimum_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::min(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "min_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        if (a != a) {
          return a;
        } else if (b != b) {
          return b;
        } else {
          return ::min(a, b);
        }
      });
    });
  }
}

void fmax_kernel_cuda(TensorIterator& iter) {
  if (isFloatingType(iter.common_dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "fmax_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::fmax(a, b);
      });
    });
  } else {
    maximum_kernel_cuda(iter);
  }
}

void fmin_kernel_cuda(TensorIterator& iter) {
  if (isFloatingType(iter.common_dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "fmin_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::fmin(a, b);
      });
    });
  } else {
    minimum_kernel_cuda(iter);
  }
}

REGISTER_DISPATCH(maximum_stub, &maximum_kernel_cuda);
REGISTER_DISPATCH(minimum_stub, &minimum_kernel_cuda);
REGISTER_DISPATCH(fmax_stub, &fmax_kernel_cuda);
REGISTER_DISPATCH(fmin_stub, &fmin_kernel_cuda);

}} // namespace at::native
