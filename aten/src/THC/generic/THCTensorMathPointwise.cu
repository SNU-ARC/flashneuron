
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPointwise.cu"
#else

#include <ATen/MemoryOverlap.h>
#include <ATen/NamedTensorUtils.h>

#if !defined(THC_REAL_IS_BOOL)

static void propagate_names_if_named_tensor_enabled(THCTensor* result, THCTensor* src) {
  at::namedinference::propagate_names(result, src);
}

void THCTensor_(crossKernel)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, nself, nx, ny, TensorCrossOp<scalar_t>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}
#endif
#endif
