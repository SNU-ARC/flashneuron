
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MultiLabelMarginCriterion.cu"
#else

static inline void THNN_(MultiLabelMarginCriterion_shapeCheck)(
  THCState *state,
  THCTensor *input, THCTensor *target) {
  int64_t ndims = input->dim();
  bool valid_inputs = (ndims == 2 && input->size(1) != 0) || (ndims == 1 && input->size(0) != 0) || ndims == 0;
  TORCH_CHECK(
    valid_inputs,
    "Expected non-empty vector or matrix with optional 0-dim batch size, but got: ",
    input->sizes());

  if (ndims <= 1) {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);

    TORCH_CHECK(valid_inputs && target->dim() <= 1 && target->numel() == dim,
      "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else if (ndims == 2) {
    int nframe = input->size(0);
    int dim = input->size(1);

    TORCH_CHECK(
      valid_inputs && target->dim() == 2 && target->size(0) == nframe && target->size(1) == dim,
      "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else {
    TORCH_CHECK(false, "Expected input of ndims <= 2, but got ndims: ", ndims);
  }
}

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           int64_t reduction)
{
  THNN_(MultiLabelMarginCriterion_shapeCheck)(state, input, target);
  if (input->numel() == 0) {
    return;
  }
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, target);

  if(input->dim() <= 1)
  {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THCTensor_(resize0d)(state, output);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->dim() == 2)
  {
    int nframe = input->size(0);
    int dim = input->size(1);
    dim3 blocks(input->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduction != at::Reduction::None)
    {
      THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size(0));
      THCTensor_(resize0d)(state, output);

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_tmp),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          reduction == at::Reduction::Mean
          );
      THCudaCheck(cudaGetLastError());
      auto t = THTensor_wrap(output_tmp);
      auto r = THTensor_wrap(output);
      at::native::sum_out(r, t, at::IntArrayRef(std::vector<int64_t>{}), false, r.scalar_type());
      THCTensor_(free)(state, output_tmp);
    }
    else
    {
      THCTensor_(resize1d)(state, output, input->size(0));

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          false
          );
      THCudaCheck(cudaGetLastError());
    }
  }
  else {
    TORCH_CHECK(false, "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ", 
      input->sizes());
  }

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradOutput,
            THCTensor *gradInput,
            THCTensor *istarget,
            int64_t reduction)
{
  THNN_(MultiLabelMarginCriterion_shapeCheck)(state, input, target);
  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);
  if (input->numel() == 0) {
    THCTensor_(free)(state, input);
    return;
  }

  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if(gradInput->dim() <= 1)
  {
    int dim = gradInput->dim() == 0 ? 1 : gradInput->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THArgCheck(!target->is_empty() && (target->dim() <= 1) && (target_size == dim), 3,
               "inconsistent target size");
    TORCH_CHECK(target->sizes() == istarget->sizes(), "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);

  }
  else if(gradInput->dim() == 2)
  {
    int nframe = gradInput->size(0);
    int dim = gradInput->size(1);
    THArgCheck((input->size(1) != 0) && (target->dim() == 2) && (target->size(0) == nframe)
               && (target->size(1) == dim), 3, "inconsistent target size");
    THArgCheck((istarget->dim() == 2) && (istarget->size(0) == nframe)
               && (istarget->size(1) == dim), 3, "inconsistent isTarget size");

    dim3 blocks(gradInput->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size(0), gradInput->size(1),
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);
  }
  else {
    TORCH_CHECK(false, "Expected 2D input with optional zero batch dim, or 1D input with non-zero dims, but got sizes: ",
      gradInput->sizes());
  }

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  THCTensor_(free)(state, gradOutput);
}

#endif
