
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/RReLU.cu"
#else

#include <THCUNN/common.h>
#include <ATen/CUDAGeneratorImpl.h>

void THNN_(RReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace,
           c10::optional<at::Generator> generator)
{
  THCUNN_assertSameGPU(state, 3, input, output, noise);
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(generator, at::cuda::detail::getDefaultCUDAGenerator());
  if (train)
  {
    auto inputTensor = THTensor_wrap(input).contiguous();
    input = inputTensor.unsafeGetTensorImpl();
    THCTensor_(resizeAs)(state, noise, input);
    scalar_t *input_data = THCTensor_(data)(state, input);
    scalar_t *noise_data = THCTensor_(data)(state, noise);
    ptrdiff_t n = THCTensor_(nElement)(state, input);

    // philox offset calculation for grid-stride loop utilizing curand4
    const uint32_t curand4_engine_calls = 4;
    dim3 grid = NUM_BLOCKS(n);
    uint64_t counter_offset = ((n - 1) / (BLOCK_SIZE * grid.x) + 1) * curand4_engine_calls;
    at::PhiloxCudaState rng_engine_inputs;
    {
      // See Note [Acquire lock when using random generators]
      std::lock_guard<std::mutex> lock(gen->mutex_);
      rng_engine_inputs = gen->philox_cuda_state(counter_offset);
    }
    if (inplace)
    {
      rreluUpdateOutputTrain<<<grid, BLOCK_SIZE, 0, c10::cuda::getCurrentCUDAStream()>>>(
        n, rng_engine_inputs, input_data, noise_data, input_data, lower, upper);
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      scalar_t *output_data = THCTensor_(data)(state, output);
      rreluUpdateOutputTrain<<<grid, BLOCK_SIZE, 0, c10::cuda::getCurrentCUDAStream()>>>(
        n, rng_engine_inputs, input_data, noise_data, output_data, lower, upper);
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    const scalar_t negSlope = ScalarConvert<double, scalar_t>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply1<scalar_t>(state, input, RReLUUpdateOutputEvalIP_functor<scalar_t>(negSlope));
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      THC_pointwiseApply2<scalar_t, scalar_t>(state, output, input, RReLUUpdateOutputEval_functor<scalar_t>(negSlope));
    }
  }
}
#endif
