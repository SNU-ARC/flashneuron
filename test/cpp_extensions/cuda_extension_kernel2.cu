#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#include <ATen/ATen.h>

__global__ void tanh_add_kernel(
    const float* __restrict__ x,
    const float* __restrict__ y,
    float* __restrict__ output,
    const int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    const float tanh_x = 2.0f / (1.0f + __expf(-2.0f * x[index])) - 1;
    const float tanh_y = 2.0f / (1.0f + __expf(-2.0f * y[index])) - 1;
    output[index] = tanh_x + tanh_y;
  }
}

void tanh_add_cuda(const float* x, const float* y, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  tanh_add_kernel<<<blocks, threads>>>(x, y, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
