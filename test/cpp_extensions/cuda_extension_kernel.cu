#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#include <ATen/ATen.h>

__global__ void sigmoid_add_kernel(
    const float* __restrict__ x,
    const float* __restrict__ y,
    float* __restrict__ output,
    const int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    const float sigmoid_x = 1.0f / (1.0f + __expf(-x[index]));
    const float sigmoid_y = 1.0f / (1.0f + __expf(-y[index]));
    output[index] = sigmoid_x + sigmoid_y;
  }
}

void sigmoid_add_cuda(const float* x, const float* y, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  sigmoid_add_kernel<<<blocks, threads>>>(x, y, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
