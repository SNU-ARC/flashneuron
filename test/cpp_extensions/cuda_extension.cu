#include "hip/hip_runtime.h"
// NOTE: This is a copy of cuda_extension_kernel.cu. It's kept here to test
// collision handling when a C++ file and CUDA file share the same filename.
// Setuptools can't deal with this at all, so the setup.py-based test uses
// cuda_extension_kernel.cu and the JIT test uses this file. Symlinks don't
// work well on Windows, so this is the most thorough solution right now.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>

#include <ATen/ATen.h>

__global__ void sigmoid_add_kernel(
    const float* __restrict__ x,
    const float* __restrict__ y,
    float* __restrict__ output,
    const int size) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    const float sigmoid_x = 1.0f / (1.0f + __expf(-x[index]));
    const float sigmoid_y = 1.0f / (1.0f + __expf(-y[index]));
    output[index] = sigmoid_x + sigmoid_y;
  }
}

void sigmoid_add_cuda(const float* x, const float* y, float* output, int size) {
  const int threads = 1024;
  const int blocks = (size + threads - 1) / threads;
  sigmoid_add_kernel<<<blocks, threads>>>(x, y, output, size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
