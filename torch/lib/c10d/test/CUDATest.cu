#include "hip/hip_runtime.h"
#include <c10d/test/CUDATest.hpp>
#include <ATen/cuda/Exceptions.h>

namespace c10d {
namespace test {

namespace {
__global__ void waitClocks(const uint64_t count) {
  clock_t start = clock64();
  clock_t offset = 0;
  while (offset < count) {
    offset = clock() - start;
  }
}

} // namespace

void cudaSleep(at::cuda::CUDAStream& stream, uint64_t clocks) {
  waitClocks<<<1, 1, 0, stream.stream()>>>(clocks);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

int cudaNumDevices() {
  int n = 0;
  C10_CUDA_CHECK_WARN(hipGetDeviceCount(&n));
  return n;
}

} // namespace test
} // namespace c10d
