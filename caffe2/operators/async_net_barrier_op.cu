#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/async_net_barrier_op.h"

namespace caffe2 {

REGISTER_HIP_OPERATOR(AsyncNetBarrier, AsyncNetBarrierOp<HIPContext>);

} // namespace caffe2
