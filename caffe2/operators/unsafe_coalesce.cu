#include "caffe2/operators/unsafe_coalesce.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

REGISTER_HIP_OPERATOR(UnsafeCoalesce, UnsafeCoalesceOp<HIPContext>);

}
